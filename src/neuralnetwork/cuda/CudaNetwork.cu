#include "hip/hip_runtime.h"

#include "CudaNetwork.hpp"
#include "Util.hpp"
#include "Random.hpp"

#include <cassert>
#include <cmath>
#include <vector>
#include <iostream>
#include <cstdio>

#include <hiprand.h>
#include <hip/hip_runtime.h>

// The default num threads per 2D block.
static constexpr int TPB_X = 16;
static constexpr int TPB_Y = 16;

using namespace neuralnetwork;
using namespace neuralnetwork::cuda;
using namespace std;

static Random rnd;
static NetworkSpec networkSpec;
static vector<LayerWeights> d_layerWeights;
static vector<LayerWeights> d_layerGradients;
static vector<LayerBatchOutputs> d_layerOutputs;
static vector<LayerBatchDeltas> d_layerDeltas;
static SamplesBatch d_samplesBatch;

// Pre-allocated all of the device memory we will need. We should never have to malloc device
// memory after this function is called.
static void allocDeviceMemory(void) {
  vector<unsigned> layerSizes(networkSpec.hiddenLayers.size() + 1);
  for (unsigned i = 0; i < networkSpec.hiddenLayers.size(); i++) {
    layerSizes[i] = networkSpec.hiddenLayers[i];
  }
  layerSizes[networkSpec.hiddenLayers.size()] = networkSpec.numOutputs;

  // This is for the input layer
  d_layerOutputs.push_back(
      util::NewLayerBatchOutputs(networkSpec.maxBatchSize, networkSpec.numInputs + 1));

  for (unsigned i = 0; i < layerSizes.size(); i++) {
    unsigned prevLayerSize = i == 0 ? networkSpec.numInputs : layerSizes[i-1];

    d_layerWeights.push_back(util::NewLayerWeights(prevLayerSize + 1, layerSizes[i]));
    d_layerGradients.push_back(util::NewLayerWeights(prevLayerSize + 1, layerSizes[i]));
    d_layerOutputs.push_back(util::NewLayerBatchOutputs(networkSpec.maxBatchSize, layerSizes[i] + 1));
    d_layerDeltas.push_back(util::NewLayerBatchDeltas(networkSpec.maxBatchSize, layerSizes[i]));
  }

  d_samplesBatch = util::NewSamplesBatch(networkSpec.maxBatchSize, networkSpec.numInputs);
}

static void freeDeviceMemory(void) {
  for (auto& lw : d_layerWeights) { util::DeleteLayerWeights(lw); }
  for (auto& lg : d_layerGradients) { util::DeleteLayerWeights(lg); }
  for (auto& lo : d_layerOutputs) { util::DeleteLayerBatchOutputs(lo); }
  for (auto& ld : d_layerDeltas) { util::DeleteLayerBatchDeltas(ld); }
  util::DeleteSamplesBatch(d_samplesBatch);
}

__global__ void initialiseLayerWeights(LayerWeights layer, const float initRange, Random rnd) {
  const unsigned row = blockDim.y * blockIdx.y + threadIdx.y;
  const unsigned col = blockDim.x * blockIdx.x + threadIdx.x;

  if (row >= layer.layerSize || col >= layer.inputSize) {
    return;
  }

  float *out = layer.Elem(row, col);
  *out = initRange * (rnd.SampleUniform(col + row * layer.inputSize) * 2.0f - 1.0f);
}

static void initialiseWeights(void) {
  for (auto& lw : d_layerWeights) {
    // Blocks per grid in X and Y dimensions.
    int bpgX = (lw.inputSize + TPB_X - 1) / TPB_X;
    int bpgY = (lw.layerSize + TPB_Y - 1) / TPB_Y;

    float initRange = 1.0f / sqrtf(lw.inputSize);
    initialiseLayerWeights<<<dim3(bpgX, bpgY, 1), dim3(TPB_X, TPB_Y, 1)>>>(lw, initRange, rnd);
  }
}

__global__ void initialiseLayerOutputs(LayerBatchOutputs outputs) {
  const unsigned id = blockDim.x * blockIdx.x + threadIdx.x;
  if (id >= outputs.batchSize) {
    return;
  }

  *(outputs.OutputElem(id, outputs.layerSize)) = 1.0f;
}

static void initialiseOutputs(void) {
  // We initialise the outputs array for each layer to have a 1.0 at the end so that it can
  // be used as the bias input for the next layer.
  for (auto& lo : d_layerOutputs) {
    int bpgX = (lo.batchSize + TPB_X - 1) / TPB_X;
    initialiseLayerOutputs<<<bpgX, TPB_X>>>(lo);
  }
}

void CudaNetwork::Initialise(const NetworkSpec &spec) {
  rnd = Random::Create(2048, 1337);

  networkSpec = spec;
  allocDeviceMemory();
  initialiseWeights();
  initialiseOutputs();
}

void CudaNetwork::Cleanup(void) {
  freeDeviceMemory();
}

void CudaNetwork::SetWeights(const std::vector<math::MatrixView> &weights) {
  assert(d_layerWeights.size() == weights.size());

  for (unsigned i = 0; i < weights.size(); i++) {
    assert(weights[i].rows == d_layerWeights[i].layerSize);
    assert(weights[i].cols == d_layerWeights[i].inputSize);

    hipError_t err = hipMemcpy2D(
        d_layerWeights[i].weights, d_layerWeights[i].pitch,
        weights[i].data, weights[i].cols * sizeof(float),
        weights[i].cols, weights[i].rows,
        hipMemcpyHostToDevice);

    CheckError(err);
  }
}

void CudaNetwork::GetWeights(std::vector<math::MatrixView> &outWeights) {
  assert(outWeights.size() == d_layerWeights.size());

  for (unsigned i = 0; i < outWeights.size(); i++) {
    assert(outWeights[i].rows == d_layerWeights[i].layerSize);
    assert(outWeights[i].cols == d_layerWeights[i].inputSize);

    hipError_t err = hipMemcpy2D(
        outWeights[i].data, outWeights[i].cols * sizeof(float), // dst
        d_layerWeights[i].weights, d_layerWeights[i].pitch, // src
        outWeights[i].cols * sizeof(float), outWeights[i].rows, // width, height
        hipMemcpyDeviceToHost);

    CheckError(err);
  }
}

static void forwardPass(const math::MatrixView &batchInputs);
static void backwardPass(const math::MatrixView &batchOutputs);

void CudaNetwork::Train(const math::MatrixView &batchInputs, const math::MatrixView &batchOutputs) {
    forwardPass(batchInputs);
}

__device__ float activationValue(float in, const LayerActivation activation) {
  switch(activation) {
  case LayerActivation::TANH:
    return tanhf(in);
  case LayerActivation::LOGISTIC:
    return 1.0f / (1.0f + expf(-in));
  case LayerActivation::RELU:
    return fmaxf(0.0f, in);
  case LayerActivation::LEAKY_RELU:
    return fmaxf(0.01f * in, in);
  case LayerActivation::LINEAR:
  case LayerActivation::SOFTMAX:
    return in;
  }
  assert(false); // should never get here.
  return in;
}

__device__ float activationDerivative(float in, float out, const LayerActivation activation) {
  switch(activation) {
  case LayerActivation::TANH:
    return 1.0f - out * out;
  case LayerActivation::LOGISTIC:
    return out * (1.0f - out);
  case LayerActivation::RELU:
    return in > 0.0f ? 1.0f : 0.0f;
  case LayerActivation::LEAKY_RELU:
    return in > 0.0f ? 1.0f : 0.01f;
  case LayerActivation::LINEAR:
  case LayerActivation::SOFTMAX:
    return 1.0f;
  }
  assert(false); // should never get here.
  return 1.0f;
}

__global__ void layerForwardPass(LayerWeights lw, LayerBatchOutputs prevOutputs,
                                 LayerBatchOutputs out, const LayerActivation activation) {
  extern __shared__ float buf[]; // shared memory buffer

  const unsigned row = blockDim.y * blockIdx.y + threadIdx.y;
  const unsigned col = blockDim.x * blockIdx.x + threadIdx.x;

  if (row >= out.batchSize || col >= out.layerSize) {
    return;
  }

  const int numChunks = (lw.inputSize + blockDim.x - 1) / blockDim.x;

  // buffer for holding the layer weight matrix chunk
  float *lwChunk = (float *) buf;

  // buffer for holding the prev outputs matrix chunk
  float *poChunk = (float *) &buf[blockDim.x * blockDim.y];

  float sum = 0.0f;
  for (int i = 0; i < numChunks; i++) {
    const int chunkOffset = i * blockDim.x;
    const int chunkIndex = threadIdx.x + threadIdx.y * blockDim.x;

    const int lwRow = col;
    const int lwCol = chunkOffset + threadIdx.x;

    const int poRow = row;
    const int poCol = lwCol;

    // if (poCol < a.cols) {
    //   lwChunk[chunkIndex] = *lw.Elem(lwRow, lwCol);
    //   poChunk[chunkIndex] = *prevOutputs.OutputElem()
    // }

    __syncthreads();

  }

  float *outElem = out.OutputElem(row, col);
  *outElem = activationValue(sum, activation);

  float *dElem = out.DerivativeElem(row, col);
  *dElem = activationDerivative(sum, *outElem, activation);
}

void forwardPass(const math::MatrixView &batchInputs) {
  // copy the batch inputs into the first layer outputs.
  hipError_t err = hipMemcpy2D(
      d_layerOutputs[0].output, d_layerOutputs[0].opitch, // dst
      batchInputs.data, batchInputs.cols * sizeof(float), // src
      batchInputs.cols * sizeof(float), batchInputs.rows, // width, height
      hipMemcpyHostToDevice);
  CheckError(err);

  for (unsigned i = 1; i < d_layerOutputs.size(); i++) {
    int bpgX = (d_layerOutputs[i].layerSize + TPB_X - 1) / TPB_X;
    int bpgY = (d_layerOutputs[i].batchSize + TPB_Y - 1) / TPB_Y;
    size_t sharedMemSize = 2 * TPB_X * TPB_Y * sizeof(float);

    LayerActivation activation = (i == d_layerOutputs.size() - 1) ?
        networkSpec.outputActivation : networkSpec.hiddenActivation;

    layerForwardPass<<<dim3(bpgX, bpgY, 1), dim3(TPB_X, TPB_Y, 1), sharedMemSize>>>(
        d_layerWeights[i-1], d_layerOutputs[i-1], d_layerOutputs[i], activation);
  }
}
