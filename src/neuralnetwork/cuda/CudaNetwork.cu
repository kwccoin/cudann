#include "hip/hip_runtime.h"

#include "CudaNetwork.hpp"
#include "Util.hpp"
#include "Random.hpp"
#include "SoftmaxKernel.hpp"
#include "ForwardPassKernel.hpp"
#include "Constants.hpp"

#include <cassert>
#include <cmath>
#include <vector>
#include <iostream>
#include <cstdio>

#include <hiprand.h>
#include <hip/hip_runtime.h>

using namespace neuralnetwork;
using namespace neuralnetwork::cuda;
using namespace std;

static Random rnd;
static NetworkSpec networkSpec;
static vector<LayerWeights> d_layerWeights;
static vector<LayerWeights> d_layerGradients;
static vector<LayerBatchOutputs> d_layerOutputs;
static vector<LayerBatchDeltas> d_layerDeltas;
static SamplesBatch d_samplesBatch;

static LayerWeights d_transposeScratch;

// Pre-allocated all of the device memory we will need. We should never have to malloc device
// memory after this function is called.
static void allocDeviceMemory(void) {
  vector<unsigned> layerSizes(networkSpec.hiddenLayers.size() + 1);
  for (unsigned i = 0; i < networkSpec.hiddenLayers.size(); i++) {
    layerSizes[i] = networkSpec.hiddenLayers[i];
  }
  layerSizes[networkSpec.hiddenLayers.size()] = networkSpec.numOutputs;

  // This is for the input layer
  d_layerOutputs.push_back(
      util::NewLayerBatchOutputs(networkSpec.maxBatchSize, networkSpec.numInputs + 1));

  unsigned maxInputSize = 0;
  unsigned maxLayerSize = 0;

  for (unsigned i = 0; i < layerSizes.size(); i++) {
    unsigned prevLayerSize = i == 0 ? networkSpec.numInputs : layerSizes[i-1];

    maxInputSize = max(maxInputSize, prevLayerSize + 1);
    maxLayerSize = max(maxLayerSize, layerSizes[i]);

    d_layerWeights.push_back(util::NewLayerWeights(prevLayerSize + 1, layerSizes[i]));
    d_layerGradients.push_back(util::NewLayerWeights(prevLayerSize + 1, layerSizes[i]));
    d_layerOutputs.push_back(util::NewLayerBatchOutputs(networkSpec.maxBatchSize, layerSizes[i] + 1));
    d_layerDeltas.push_back(util::NewLayerBatchDeltas(networkSpec.maxBatchSize, layerSizes[i]));
  }

  d_samplesBatch = util::NewSamplesBatch(networkSpec.maxBatchSize, networkSpec.numInputs);
  d_transposeScratch = util::NewLayerWeights(maxLayerSize, maxInputSize);
}

static void freeDeviceMemory(void) {
  for (auto& lw : d_layerWeights) { util::DeleteLayerWeights(lw); }
  for (auto& lg : d_layerGradients) { util::DeleteLayerWeights(lg); }
  for (auto& lo : d_layerOutputs) { util::DeleteLayerBatchOutputs(lo); }
  for (auto& ld : d_layerDeltas) { util::DeleteLayerBatchDeltas(ld); }
  util::DeleteSamplesBatch(d_samplesBatch);
  util::DeleteLayerWeights(d_transposeScratch);
}

__global__ void initialiseLayerWeights(LayerWeights layer, const float initRange, Random rnd) {
  const unsigned row = blockDim.y * blockIdx.y + threadIdx.y;
  const unsigned col = blockDim.x * blockIdx.x + threadIdx.x;

  if (row >= layer.layerSize || col >= layer.inputSize) {
    return;
  }

  float *out = layer.Elem(row, col);
  *out = row + col; //initRange * (rnd.SampleUniform(col + row * layer.inputSize) * 2.0f - 1.0f);
}

static void initialiseWeights(void) {
  for (auto& lw : d_layerWeights) {
    // Blocks per grid in X and Y dimensions.
    int bpgX = (lw.inputSize + TPB_X - 1) / TPB_X;
    int bpgY = (lw.layerSize + TPB_Y - 1) / TPB_Y;

    float initRange = 1.0f / sqrtf(lw.inputSize);
    initialiseLayerWeights<<<dim3(bpgX, bpgY, 1), dim3(TPB_X, TPB_Y, 1)>>>(lw, initRange, rnd);
  }
}

__global__ void initialiseLayerOutputs(LayerBatchOutputs outputs) {
  const unsigned id = blockDim.x * blockIdx.x + threadIdx.x;
  if (id >= outputs.maxBatchSize) {
    return;
  }

  *(outputs.OutputElem(id, outputs.layerSize - 1)) = 1.0f;
}

static void initialiseOutputs(void) {
  // We initialise the outputs array for each layer to have a 1.0 at the end so that it can
  // be used as the bias input for the next layer.
  for (auto& lo : d_layerOutputs) {
    int bpgX = (lo.maxBatchSize + TPB_X - 1) / TPB_X;
    initialiseLayerOutputs<<<bpgX, TPB_X>>>(lo);
  }
}

void CudaNetwork::Initialise(const NetworkSpec &spec) {
  rnd = Random::Create(2048, 1337);

  networkSpec = spec;
  assert(networkSpec.hiddenActivation != LayerActivation::SOFTMAX);

  allocDeviceMemory();
  initialiseWeights();
  initialiseOutputs();
}

void CudaNetwork::Cleanup(void) {
  freeDeviceMemory();
}

void CudaNetwork::SetWeights(const std::vector<math::MatrixView> &weights) {
  assert(d_layerWeights.size() == weights.size());

  for (unsigned i = 0; i < weights.size(); i++) {
    assert(weights[i].rows == d_layerWeights[i].layerSize);
    assert(weights[i].cols == d_layerWeights[i].inputSize);

    hipError_t err = hipMemcpy2D(
        d_layerWeights[i].weights, d_layerWeights[i].pitch,
        weights[i].data, weights[i].cols * sizeof(float),
        weights[i].cols, weights[i].rows,
        hipMemcpyHostToDevice);

    CheckError(err);
  }
}

void CudaNetwork::GetWeights(std::vector<math::MatrixView> &outWeights) {
  assert(outWeights.size() == d_layerWeights.size());

  for (unsigned i = 0; i < outWeights.size(); i++) {
    assert(outWeights[i].rows == d_layerWeights[i].layerSize);
    assert(outWeights[i].cols == d_layerWeights[i].inputSize);

    hipError_t err = hipMemcpy2D(
        outWeights[i].data, outWeights[i].cols * sizeof(float), // dst
        d_layerWeights[i].weights, d_layerWeights[i].pitch, // src
        outWeights[i].cols * sizeof(float), outWeights[i].rows, // width, height
        hipMemcpyDeviceToHost);

    CheckError(err);
  }
}

static void forwardPass(const math::MatrixView &batchInputs);
// static void backwardPass(const math::MatrixView &batchOutputs);

void CudaNetwork::Train(const math::MatrixView &batchInputs, const math::MatrixView &batchOutputs) {
    forwardPass(batchInputs);
}

void forwardPass(const math::MatrixView &batchInputs) {
  for (auto& lo : d_layerOutputs) {
    assert(batchInputs.rows <= lo.maxBatchSize);
    lo.batchSize = batchInputs.rows;
  }

  // copy the batch inputs into the first layer outputs.
  hipError_t err = hipMemcpy2D(
      d_layerOutputs[0].output, d_layerOutputs[0].opitch, // dst
      batchInputs.data, batchInputs.cols * sizeof(float), // src
      batchInputs.cols * sizeof(float), batchInputs.rows, // width, height
      hipMemcpyHostToDevice);
  CheckError(err);

  for (unsigned i = 1; i < d_layerOutputs.size(); i++) {
    LayerActivation activation = (i == d_layerOutputs.size() - 1) ?
        networkSpec.outputActivation : networkSpec.hiddenActivation;

    ForwardPassKernel::Apply(d_layerWeights[i-1], d_layerOutputs[i-1], d_layerOutputs[i], activation);
  }

  LayerBatchOutputs lastLayer = d_layerOutputs[d_layerOutputs.size() - 1];
  if (networkSpec.outputActivation == LayerActivation::SOFTMAX) {
    SoftmaxKernel::Apply(lastLayer);
  }

  math::MatrixView output = math::MatrixView::Create(lastLayer.batchSize, lastLayer.layerSize);

  err = hipMemcpy2D(
      output.data, output.cols * sizeof(float), // dst
      lastLayer.output, lastLayer.opitch, // src
      output.cols * sizeof(float), output.rows, // width, height
      hipMemcpyDeviceToHost);
  CheckError(err);

  for (unsigned r = 0; r < output.rows; r++) {
    for (unsigned c = 0; c < output.cols; c++) {
      cout << output.data[c + r * output.cols] << "\t";
    }
    cout << endl;
  }
  cout << endl;
}
